#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__global__ void csr_dot_csc_cuda_kernel(
    const int64_t* __restrict__ t1_indices,
    const int64_t* __restrict__ t1_indptr,
    const scalar_t* __restrict__ t1_data,
    const int64_t* __restrict__ t2_indices,
    const int64_t* __restrict__ t2_indptr,
    const scalar_t* __restrict__ t2_data,
    scalar_t* __restrict__ out_dense,
    const int64_t out_h,
    const int64_t out_w
)
{
    const int64_t ij = blockIdx.x * blockDim.x + threadIdx.x;
    const int64_t b = blockIdx.y;

    if (ij < out_h * out_w)
    {
        const int64_t i = ij / out_w;
        const int64_t j = ij % out_w;

        const int64_t t1_start = t1_indptr[b * out_h + i];
        const int64_t t1_stop = t1_indptr[b * out_h + i + 1];

        const int64_t t2_start = t2_indptr[b * out_w + j];
        const int64_t t2_stop = t2_indptr[b * out_w + j + 1];

        scalar_t outp = 0;
        int64_t t1_ptr_idx = t1_start;
        int64_t t2_ptr_idx = t2_start;

        while (t1_ptr_idx < t1_stop && t2_ptr_idx < t2_stop)
        {
            int64_t t1_cur_indice = t1_indices[t1_ptr_idx];
            int64_t t2_cur_indice = t2_indices[t2_ptr_idx];
            if (t1_cur_indice == t2_cur_indice)
            {
                outp += t1_data[t1_ptr_idx] * t2_data[t2_ptr_idx];
                t1_ptr_idx++;
                t2_ptr_idx++;
            }
            else if (t1_cur_indice < t2_cur_indice)
                t1_ptr_idx++;
            else
                t2_ptr_idx++;
        }
        out_dense[b * out_w * out_h + i * out_w + j] = outp;
    }
}


at::Tensor csr_dot_csc_cuda(
    at::Tensor t1_indices,
    at::Tensor t1_indptr,
    at::Tensor t1_data,
    at::Tensor t2_indices,
    at::Tensor t2_indptr,
    at::Tensor t2_data,
    int64_t batch_size,
    int64_t out_h,
    int64_t out_w
){
    auto out_dense = at::zeros({batch_size, out_h, out_w}, t1_data.type());

    const int threads = 1024;
    const dim3 blocks((out_h * out_w + threads - 1) / threads, batch_size);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(t1_data.type(), "csr_dot_csc_cuda", ([&] {
    csr_dot_csc_cuda_kernel<scalar_t><<<blocks, threads>>>(
        t1_indices.data<int64_t>(),
        t1_indptr.data<int64_t>(),
        t1_data.data<scalar_t>(),
        t2_indices.data<int64_t>(),
        t2_indptr.data<int64_t>(),
        t2_data.data<scalar_t>(),
        out_dense.data<scalar_t>(),
        out_h,
        out_w);
    }));
    return out_dense;
}
